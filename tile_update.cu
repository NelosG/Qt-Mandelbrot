#include "hip/hip_runtime.h"
#include "tile_update.h"

__device__ std::uint8_t mand(double c_real,  double c_imag) noexcept {
    double z_real = 0;
    double z_imag = 0;
    for (int i = 0; i < 255; i++) {
        auto z_real_2 = z_real * z_real;
        auto z_imag_2 = z_imag * z_imag;
        auto z_real_imag = z_real * z_imag;

        if (z_real_2 + z_imag_2 >= 4) {
            return i;
        }

        z_real = z_real_2 - z_imag_2 + c_real;
        z_imag = z_real_imag * 2 + c_imag;
    }
    return 0;
}

__global__ void do_update_cuda(std::uint8_t* data_ptr, int h, int w, int y,
                               int bytesPerLine,
                               double diaganal_real,
                               double diaganal_imag,
                               double cor_real,
                               double cor_imag
) {
    auto curY = y + threadIdx.x;

    std::uint8_t* data = data_ptr + curY * bytesPerLine;
    auto yy = (double)curY / h * diaganal_imag + cor_imag;
    for (int x = 0; x < w; x++) {
        auto xx = (double)x / w * diaganal_real + cor_real;
        std::uint8_t val = mand(xx, yy);
        data[x * 3 + 0] = 0;
        data[x * 3 + 1] = val;
        data[x * 3 + 2] = val;
    }
}

void do_update(std::uint8_t* data_ptr, int h, int w, int y,
               int bytesPerLine,
               const std::complex<long double>& diaganal,
               const std::complex<long double>& cor) {
    auto bytes = h * bytesPerLine;
    std::uint8_t* p;
    // auto* p = (std::uint8_t*)malloc(sizeof(std::uint8_t) * bytes);
    hipMalloc((void**)&p, sizeof(std::uint8_t) * bytes);

    do_update_cuda<<<1, h - y>>>(p,
                                 h, w, y,
                                 bytesPerLine,
                                 diaganal.real(),
                                 diaganal.imag(),
                                 cor.real(),
                                 cor.imag()
    );
    hipMemcpy(data_ptr, p, sizeof(std::uint8_t) * bytes, hipMemcpyDeviceToHost);
    hipFree(p);
}
